
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>

__global__ void addKernel(int *a, int *b, int *c, long int length)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = x * length + y;
    if (x < length && y < length)
        c[index] = a[x] + b[y];
}

int main(int argc, char **argv)
{
    // initialize host memory
    if (argc == 1) {
        fprintf(stderr, "Call program with size argument\n");
        exit(EXIT_FAILURE);
    }
    long int N;
    if (sscanf(argv[1], "%ld", &N) == 0) {
        fprintf(stderr, "Size argument should be a long int\n");
        exit(EXIT_FAILURE);
    }
    int *a = (int*) malloc(sizeof(int) * N);
    int *b = (int*) malloc(sizeof(int) * N);
    int *results = (int*) malloc(sizeof(int) * N);
    int i;

    // initialize device memory
    int *dev_a;
    int *dev_b;
    int *dev_results;
    hipMalloc((void**) &dev_a, N * sizeof(int));
    hipMalloc((void**) &dev_b, N * sizeof(int));
    hipMalloc((void**) &dev_results, N * sizeof(int));

    // initialize arrays
    for (i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    // copy to device for computation
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // do add operation
    addKernel<<<N * N / 512, 512>>>(dev_a, dev_b, dev_results, N);

    hipMemcpy(results, dev_results, N * sizeof(int), hipMemcpyDeviceToHost);

    // clean up
    free(a);
    free(b);
    free(results);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_results);

    // done
    printf("done\n");
    return EXIT_SUCCESS;
}

